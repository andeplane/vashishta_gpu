#include "hip/hip_runtime.h"
// **************************************************************************
//                                   sw.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the sw pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Tue March 26, 2013
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#include <cstdio>

#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> sw1_tex;
texture<float4> sw2_tex;
texture<float4> sw3_tex;
texture<float4> sw4_tex;
texture<float4> sw5_tex;
#else
texture<int4,1> pos_tex;
texture<int4> sw1_tex;
texture<int4> sw2_tex;
texture<int4> sw3_tex;
texture<int4> sw4_tex;
texture<int4> sw5_tex;
#endif

#else
#define pos_tex x_
#define sw1_tex sw1
#define sw2_tex sw2
#define sw3_tex sw3
#define sw3_tex sw4
#define sw3_tex sw5
#endif

#define THIRD (numtyp)0.66666666666666666667

//#define THREE_CONCURRENT

#if (ARCH < 300)

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_ELLIPSE];                               \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#else

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        energy += shfl_xor(energy, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    int ei=ii;                                                              \
    if (eflag>0) {                                                          \
      engv[ei]+=energy*(acctyp)0.5;                                         \
      ei+=inum;                                                             \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        engv[ei]+=virial[i]*(acctyp)0.5;                                    \
        ei+=inum;                                                           \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#endif


__kernel void k_vashishta(const __global numtyp4 *restrict x_,
                   const __global numtyp4 *restrict sw1,
                   const __global numtyp4 *restrict sw2,
                   const __global numtyp4 *restrict sw3,
                   const __global numtyp4 *restrict sw4,
                   const __global numtyp4 *restrict sw5,
                   const __global int *restrict map,
                   const __global int *restrict elem2param,
                   const int nelements,
                   const __global int * dev_nbor,
                   const __global int * dev_packed,
                   __global acctyp4 *restrict ans,
                   __global acctyp *restrict engv,
                   const int eflag, const int vflag, const int inum,
                   const int nbor_pitch, const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<sw4[ijparam].z) { // cutsq = sw4[ijparam].z
        numtyp4 sw1_ijparam; fetch4(sw1_ijparam,ijparam,sw1_tex);
        numtyp sw1_eta=sw1_ijparam.x;
        numtyp sw1_lam1inv=sw1_ijparam.y;
        numtyp sw1_lam4inv=sw1_ijparam.z;
        numtyp sw1_zizj=sw1_ijparam.w;

        numtyp4 sw2_ijparam; fetch4(sw2_ijparam,ijparam,sw2_tex);
        numtyp sw2_mbigd=sw2_ijparam.x;
        numtyp sw2_dvrc =sw2_ijparam.y;
        numtyp sw2_big6w=sw2_ijparam.z;
        numtyp sw2_heta =sw2_ijparam.w;

        numtyp4 sw3_ijparam; fetch4(sw3_ijparam,ijparam,sw3_tex);
        numtyp sw3_bigh=sw3_ijparam.x;
        numtyp sw3_bigw=sw3_ijparam.y;
        numtyp sw3_dvrc=sw3_ijparam.z;
        numtyp sw3_c0  =sw3_ijparam.w;

        numtyp r=sqrt(rsq);
        numtyp rinvsq=1.0/rsq;
        numtyp r4inv = rinvsq*rinvsq;
        numtyp r6inv = rinvsq*r4inv;

        numtyp reta = pow(r,-sw1_eta);
        numtyp lam1r = r*sw1_lam1inv;
        numtyp lam4r = r*sw1_lam4inv;
        numtyp vc2 = sw1_zizj * exp(-lam1r)/r;
        numtyp vc3 = sw2_mbigd * r4inv*exp(-lam4r);

        numtyp force = (sw2_dvrc*r
            - (4.0*vc3 + lam4r*vc3+sw2_big6w*r6inv
               - sw2_heta*reta - vc2 - lam1r*vc2)
            ) * rinvsq;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;
        if (eflag>0)
          energy += (sw3_bigh*reta+vc2-vc3-sw3_bigw*r6inv-r*sw3_dvrc+sw3_c0);
          
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }
    } // for nbor

    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii

}

#define threebody(delr1x, delr1y, delr1z, eflag, energy)                     \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - sw_r0_ij);                                  \
  numtyp gsrainv1 = sw_gamma_ij * rainv1;                                    \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rinvsq2 = ucl_recip(rsq2);                                          \
  numtyp rainv2 = ucl_recip(r2 - sw_r0_ik);                                  \
  numtyp gsrainv2 = sw_gamma_ik * rainv2;                                    \
  numtyp gsrainvsq2 = gsrainv2*rainv2/r2;                                    \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - sw_costheta_ijk;                                       \
  numtyp delcssq = delcs*delcs;                                              \
  numtyp pcsinv = sw_bigc_ijk*delcssq+1.0;                                   \
  numtyp pcsinvsq = pcsinv*pcsinv;                                           \
  numtyp pcs = delcssq/pcsinv;                                               \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = sw_bigb_ijk * facexp*pcs;                                  \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp frad2 = facrad*gsrainvsq2;                                          \
  numtyp facang = sw_big2b_ijk * facexp*delcs/pcsinvsq;                      \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
                                                                             \
  numtyp csfac2 = rinvsq2*csfacang;                                          \
                                                                             \
  fkx = delr2x*(frad2+csfac2)-delr1x*facang12;                               \
  fky = delr2y*(frad2+csfac2)-delr1y*facang12;                               \
  fkz = delr2z*(frad2+csfac2)-delr1z*facang12;                               \
                                                                             \
  if (eflag>0)                                                               \
    energy+=facrad;                                                          \
  if (vflag>0) {                                                             \
    virial[0] += delr1x*fjx + delr2x*fkx;                                    \
    virial[1] += delr1y*fjy + delr2y*fky;                                    \
    virial[2] += delr1z*fjz + delr2z*fkz;                                    \
    virial[3] += delr1x*fjy + delr2x*fky;                                    \
    virial[4] += delr1x*fjz + delr2x*fkz;                                    \
    virial[5] += delr1y*fjz + delr2y*fkz;                                    \
  }                                                                          \
}

#define threebody_half(delr1x, delr1y, delr1z)                               \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - sw_r0_ij);                                  \
  numtyp gsrainv1 = sw_gamma_ij * rainv1;                                    \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rainv2 = ucl_recip(r2 - sw_r0_ik);                                  \
  numtyp gsrainv2 = sw_gamma_ik * rainv2;                                    \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - sw_costheta_ijk;                                       \
  numtyp delcssq = delcs*delcs;                                              \
  numtyp pcsinv = sw_bigc_ijk*delcssq+1.0;                                   \
  numtyp pcsinvsq = pcsinv*pcsinv;                                           \
  numtyp pcs = delcssq/pcsinv;                                               \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = sw_bigb_ijk * facexp*pcs;                                  \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp facang = sw_big2b_ijk * facexp*delcs/pcsinvsq;                      \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
}

__kernel void k_vashishta_three_center(const __global numtyp4 *restrict x_,
                                const __global numtyp4 *restrict sw1,
                                const __global numtyp4 *restrict sw2,
                                const __global numtyp4 *restrict sw3,
                                const __global numtyp4 *restrict sw4,
                                const __global numtyp4 *restrict sw5,
                                const __global int *restrict map,
                                const __global int *restrict elem2param,
                                const int nelements,
                                const __global int * dev_nbor,
                                const __global int * dev_packed,
                                __global acctyp4 *restrict ans,
                                __global acctyp *restrict engv,
                                const int eflag, const int vflag,
                                const int inum,  const int nbor_pitch,
                                const int t_per_atom, const int evatom) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp sw_gamma_ij, sw_r0_ij, sw_gamma_ik, sw_r0_ik;
  numtyp sw_costheta_ijk, sw_bigc_ijk, sw_bigb_ijk, sw_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end;

    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {

      int j=dev_packed[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = jx.x-ix.x;
      numtyp delr1y = jx.y-ix.y;
      numtyp delr1z = jx.z-ix.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      
      numtyp4 sw4_ijparam; fetch4(sw4_ijparam,ijparam,sw4_tex);
      sw_r0_ij=sw4_ijparam.x;
      if (rsq1 > sw_r0_ij*sw_r0_ij) continue;
      sw_gamma_ij=sw4_ijparam.y;
      
      int nbor_k=nbor_j-offset_j+offset_k;
      if (nbor_k<=nbor_j)
        nbor_k+=n_stride;

      for ( ; nbor_k<nbor_end; nbor_k+=n_stride) {
        int k=dev_packed[nbor_k];
        k &= NEIGHMASK;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[itype*nelements*nelements+ktype*nelements+ktype];
        numtyp4 sw4_ikparam; fetch4(sw4_ikparam,ikparam,sw4_tex);

        numtyp delr2x = kx.x-ix.x;
        numtyp delr2y = kx.y-ix.y;
        numtyp delr2z = kx.z-ix.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        if (rsq2 < sw4_ikparam.x*sw4_ikparam.x) {
          numtyp4 sw4_ikparam; fetch4(sw4_ikparam,ikparam,sw4_tex);
          sw_gamma_ik=sw4_ikparam.y;
          sw_r0_ik=sw4_ikparam.x;

          int ijkparam=elem2param[itype*nelements*nelements+jtype*nelements+ktype];
          numtyp4 sw5_ijkparam; fetch4(sw5_ijkparam,ijkparam,sw5_tex);
          sw_bigc_ijk=sw5_ijkparam.x;
          sw_bigb_ijk=sw5_ijkparam.z;
          sw_big2b_ijk=sw5_ijkparam.w;
          sw_costheta_ijk=sw5_ijkparam.y;

          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x -= fjx + fkx;
          f.y -= fjy + fky;
          f.z -= fjz + fkz;
        }
      }
    } // for nbor

    numtyp pre;
    if (evatom==1)
      pre=THIRD;
    else
      pre=(numtyp)2.0;
    energy*=pre;
    for (int i=0; i<6; i++)
      virial[i]*=pre;

    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);

  } // if ii
}

__kernel void k_vashishta_three_end(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict sw1,
                             const __global numtyp4 *restrict sw2,
                             const __global numtyp4 *restrict sw3,
                             const __global numtyp4 *restrict sw4,
                             const __global numtyp4 *restrict sw5,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_packed,
                             const __global int * dev_acc,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp sw_gamma_ij, sw_r0_ij, sw_gamma_ik, sw_r0_ik;
  numtyp sw_costheta_ijk, sw_bigc_ijk, sw_bigb_ijk, sw_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;

    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=dev_packed[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 sw4_ijparam; fetch4(sw4_ijparam,ijparam,sw4_tex);
      sw_r0_ij = sw4_ijparam.x;
      if (rsq1 > sw_r0_ij*sw_r0_ij) continue;

      sw_gamma_ij=sw4_ijparam.y;
      
      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_acc[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_acc[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=dev_packed[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; //jk

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        numtyp4 sw4_ikparam; fetch4(sw4_ikparam,ikparam,sw4_tex);
        sw_r0_ik=sw4_ikparam.x;

        if (rsq2 < sw_r0_ik*sw_r0_ik) {
          sw_gamma_ik=sw4_ikparam.y;
          
          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; //jik
          numtyp4 sw5_ijkparam; fetch4(sw5_ijkparam,ijkparam,sw5_tex);
          sw_bigc_ijk=sw5_ijkparam.x;
          sw_costheta_ijk=sw5_ijkparam.y;
          sw_bigb_ijk=sw5_ijkparam.z;
          sw_big2b_ijk=sw5_ijkparam.w;
          
          numtyp fjx, fjy, fjz;
          //if (evatom==0) {
            threebody_half(delr1x,delr1y,delr1z);
          //} else {
          //  numtyp fkx, fky, fkz;
          //  threebody(delr1x,delr1y,delr1z,eflag,energy);
          //}

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

__kernel void k_vashishta_three_end_vatom(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict sw1,
                             const __global numtyp4 *restrict sw2,
                             const __global numtyp4 *restrict sw3,
                             const __global numtyp4 *restrict sw4,
                             const __global numtyp4 *restrict sw5,
                             const __global int *restrict map,
                             const __global int *restrict elem2param,
                             const int nelements,
                             const __global int * dev_nbor,
                             const __global int * dev_packed,
                             const __global int * dev_acc,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             const int eflag, const int vflag,
                             const int inum,  const int nbor_pitch,
                             const int t_per_atom, const int gpu_nbor) {
  __local int tpa_sq, n_stride;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  numtyp sw_gamma_ij, sw_r0_ij, sw_gamma_ik, sw_r0_ik;
  numtyp sw_costheta_ijk, sw_bigc_ijk, sw_bigb_ijk, sw_big2b_ijk;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor_j, nbor_end, k_end;

    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,nbor_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    itype=map[itype];

    for ( ; nbor_j<nbor_end; nbor_j+=n_stride) {
      int j=dev_packed[nbor_j];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      jtype=map[jtype];

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;

      int ijparam=elem2param[itype*nelements*nelements+jtype*nelements+jtype];
      numtyp4 sw4_ijparam; fetch4(sw4_ijparam,ijparam,sw4_tex);
      sw_r0_ij=sw4_ijparam.x;
      if (rsq1 > sw_r0_ij*sw_r0_ij) continue;

      sw_gamma_ij=sw4_ijparam.y;
      
      int nbor_k,numk;
      if (dev_nbor==dev_packed) {
        if (gpu_nbor) nbor_k=j+nbor_pitch;
        else nbor_k=dev_acc[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k=dev_acc[j]+nbor_pitch;
        numk=dev_nbor[nbor_k];
        nbor_k+=nbor_pitch;
        nbor_k=dev_nbor[nbor_k];
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=dev_packed[nbor_k];
        k &= NEIGHMASK;

        if (k == i) continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        int ktype=kx.w;
        ktype=map[ktype];
        int ikparam=elem2param[jtype*nelements*nelements+ktype*nelements+ktype]; // jk
        numtyp4 sw4_ikparam; fetch4(sw4_ikparam,ikparam,sw4_tex);

        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        sw_r0_ik=sw4_ikparam.x;

        if (rsq2 < sw_r0_ik*sw_r0_ik) {
          sw_gamma_ik=sw4_ikparam.y;

          int ijkparam=elem2param[jtype*nelements*nelements+itype*nelements+ktype]; // jik
          numtyp4 sw5_ijkparam; fetch4(sw5_ijkparam,ijkparam,sw5_tex);
          sw_bigc_ijk=sw5_ijkparam.x;
          sw_costheta_ijk=sw5_ijkparam.y;
          sw_bigb_ijk=sw5_ijkparam.z;
          sw_big2b_ijk=sw5_ijkparam.w;
          
          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    energy*=THIRD;
    for (int i=0; i<6; i++)
      virial[i]*=THIRD;
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

